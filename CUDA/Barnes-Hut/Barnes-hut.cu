
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) getchar();
   }
}

char fileInput[] = "../../Generate/particle.txt";
double *x,*y,*m,*velX,*velY,*forceX,*forceY;
double maxSize = 6.162025e+070;
int numberBody, seed, maxTime = 3;

__constant__ double G = 6.67384E-11; // costante gravitazione universale
__constant__ double THETA = 0.5; // thetha per il calcolo delle forze su particell
__device__ int ppointer;


// double maxSize = 100;
// int count = 0;

 //&p1[i].x, &p1[i].y, &p1[i].mass, &p1[i].velX, &p1[i].velY



__device__ int findCell(int x,int y){
    printf("ppointer:%d\n",ppointer);
}

__global__ void createTree(double* xP,double* yP,double* up,double* down,double* left,double* right,int* child){
    
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    int cell=findCell(xP[id],yP[id]);
}

__global__ void setppointer(int num){
    ppointer=num;
}

int statGPU() {
    int numberGPU;
    hipGetDeviceCount(&numberGPU);
    if(numberGPU<1){
        printf("non sono state rilevate GPU adeguate per esegiure il programma");
        exit(1);
    }

    hipDeviceProp_t pr;
    hipGetDeviceProperties(&pr,0);//thread per blocco 877
    int f = pr.sharedMemPerBlock/sizeof(double); //massima dim memoria per blocco/grandezza struct particella 
    //printf("\n%d\n",f);

    if(pr.maxThreadsPerMultiProcessor%f){

        int h=pr.maxThreadsPerMultiProcessor;

        while (h>f)
        {
            h=h/2;
        }
        
        f=h;
    }
    //printf("\n%d\n",f);
    return f;
}

/*void printerFile(particle *p1)
{
    FILE* solution=fopen("solution.txt","w");
    for (int i = 0; i < numberBody; i++)
    {
        fprintf(solution,"%e,%e,%e,%e,%e,%e,%e\n", p1[i].x, p1[i].y, p1[i].mass, p1[i].forceX, p1[i].forceY, p1[i].velX, p1[i].velY);
    }
    fclose(solution);
}*/

void printer()
{
    for (int i = 0; i < numberBody; i++)
    {
        printf("particle xPos= %e, yPos= %e, mass= %e\n", x[i], y[i], m[i]);//, forceX, forceY, velX, velY); , forceX= %e, forceY= %e, velX= %e, velY= %e
    }
}

// calcolo il movimento delle particelle nel tempo richiesto
void compute(int time)
{
    /*
    int thread=statGPU();
    int block=(numberBody/thread)+1;
    */
    //int sizeTree=numberBody*2+12000;

    double *xP,*yP,*up,*down,*left,*right;
    int *child;
    
    // allocazione della memoria a device
    // gpuErrchk(); da aggiungere
    //cudaGetLastError
    printf("ciao\n");
    
    gpuErrchk(hipMalloc((void**)&xP,sizeof(double) * numberBody));
    hipMalloc((void**)&yP,sizeof(double) * numberBody);
    hipMalloc((void**)&up,sizeof(double));
    hipMalloc((void**)&down,sizeof(double));
    hipMalloc((void**)&left,sizeof(double));
    hipMalloc((void**)&right,sizeof(double));
    hipMalloc((void**)&child,sizeof(int)*(numberBody*2+12000)*4);    
    
    hipMemcpy(xP,x,sizeof(double) * numberBody,hipMemcpyHostToDevice);
    hipMemcpy(yP,y,sizeof(double) * numberBody,hipMemcpyHostToDevice);
    hipMemset(up,maxSize,sizeof(double));
    hipMemset(down,-maxSize,sizeof(double));
    hipMemset(left,-maxSize,sizeof(double));
    hipMemset(right,maxSize,sizeof(double));
    hipMemset(&child[((numberBody*2+12000)*4)-1],-1,sizeof(int));

    setppointer<<<1,1>>>(((numberBody*2+12000)*4)-1);
    hipDeviceSynchronize();
    
    for(int i=0;i<time;i++){
        
        createTree<<<4,1>>>(xP,yP,up,down,left,right,child);
        hipDeviceSynchronize();
        //calculateCenterMass<<<?>>>(?);
        //cudaDeviceSynchronize();
        //calculateMove<<<?>>>(?);
        //cudaDeviceSynchronize();
                                                                                            //printf("\ncambio\n");
    }
    hipFree(xP);
    hipFree(yP);
    hipFree(up);
    hipFree(down);
    hipFree(left);
    hipFree(right);
    hipFree(child);

}

// popolo l'array con le particelle nel file
void getInput(FILE *file)
{
    x= (double*) malloc(sizeof(double)*numberBody);
    y= (double*) malloc(sizeof(double)*numberBody);
    m= (double*) malloc(sizeof(double)*numberBody);
    
    velX= (double*) malloc(sizeof(double)*numberBody);
    velY= (double*) malloc(sizeof(double)*numberBody);
    forceX= (double*) malloc(sizeof(double)*numberBody);
    forceY= (double*) malloc(sizeof(double)*numberBody);
    // prendo i dati per tutti i corpi
    for (int i = 0; i < numberBody; i++)
    {   
        // prendo i dati dal file
        fscanf(file, "%lf%lf%lf%lf%lf", &x[i], &y[i], &m[i], &velX[i], &velY[i]);
        // imposto le forze iniziali a zero
        forceX[i]=0;
        forceY[i]=0;
        //printf("particle xPos= %e, yPos= %e, mass= %e, forceX= %e, forceY= %e, velX= %e, velY= %e\n", x[i], y[i], m[i], forceX[i], forceY[i], velX[i], velY[i]);
    }
    // chiudo il file
    fclose(file);
}

// aprire il file e prendere i primi valori (seed e numero di corpi)
FILE *initial()
{
    // mi apro il file in lettura
    FILE *file = fopen(fileInput, "r");
    // prendo il seed
    fscanf(file, "%d", &seed);
    printf("%d\n", seed);
    // prendo il numero di corpi
    fscanf(file, "%d", &numberBody);
    printf("%d\n", numberBody);
    return file;
}

int main()
{
    // apro il file dove si trovano tutte le particelle
    FILE *file = initial();
    // alloco memoria per variabili host
    //inizializzo array di indirizzi child
    // popolo gli array
    getInput(file);

    // calcolo il movimento delle particelle nel tempo richiesto
    compute(maxTime);
    printf("\n");
    //printer(p1);

    //printerFile(p1);
    fclose(file);
    exit(1);
}