
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>

// costanti e variabili host
int maxCells, numberBody, seed, maxTime = 1;
char fileInput[] = "../../Generate/particle.txt";
double *x, *y, *m, *velX, *velY, *forceX, *forceY;
//double maxSize=50;
double maxSize = 6.162025e+070;

// costanti e variabili gpu
__constant__ double G = 6.67384E-11; // costante gravitazione universale
__constant__ double THETA = 0.5;     // thetha per il calcolo delle forze su particell
__device__ int pPointer;

///////////////////////////////////////////GPU ERRORCHECK///////////////////////////////////////////////////////////////
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}
__device__ int h = 0;
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// funzioni gpu

// funzione kernel per trovare la particella da inserire
__global__ void createTree(double* x, double* y, double up, double down, double left, double right, int *child,int cell,int numBody)
{   
    int body = threadIdx.x + blockDim.x * blockIdx.x;
    
    // uccido il thread che non deve inserire particelle
    if(body>numBody){
        return;
    }
                                                numBody+=1;
    int father=cell;
    bool newBody=true;
    bool finish=false;
    int childPath;
    while(!finish){
        if(newBody){
            newBody = false;
            childPath = 0;
            //    _________________________
            //   |          3 |          1 |
            //   |    (NW)    |    (NE)    |
            //   |            |            |
            //   |     -+     |     ++     |
            //   |____________|____________|
            //   |          2 |          0 |
            //   |     --     |     +-     |
            //   |            |            |
            //   |    (SW)    |    (SE)    |
            //   |____________|____________|
            //
            if(x[body]<=0.5*(left+right)){
                //+2
                childPath +=2;
                right = 0.5*(left+right);
            } else {
                //+0
                left = 0.5*(left+right);
            }
            if(y[body]>0.5*(up+down)){
                //+1
                childPath +=1;
                down = 0.5*(up+down);
            }else{
                //+0
                up = 0.5*(up+down);
            }  
        }
        cell=child[father-childPath];

        // ciclo fino a che non trovo una foglia
        while(cell >= numBody){
            
            father = cell;
            childPath=0;

            if(x[body]<=0.5*(left+right)){
                //+2
                childPath +=2;
                right = 0.5*(left+right);
            } else {
                //+0
                left = 0.5*(left+right);
            }
            if(y[body]>0.5*(up+down)){
                //+1
                childPath +=1;
                down = 0.5*(up+down);
            }else{
                //+0
                up = 0.5*(up+down);
            }  

            //Possibbile creazione di centro di massa
            cell = child[father - childPath];
        }
                                                                                //printf("cell: %d\n",cell);
        if (cell != -2){
            int lock=father-childPath;
                                                                                //printf("cell2: %d\n",cell);
            if(atomicCAS(&child[lock],cell,-2)==cell){
                if(cell == -1){
                                                                                printf("lock:%d id:%d d %f, u %f, r %f, l %f\n",lock,body,down,up,right,left);
                    child[lock] = body;
                    finish=true;
                    
                }else{
                    while(cell>=0 && cell<numBody){
                        int newCell = atomicAdd(&pPointer,-4);

                        //possibilità di omettere
                        child[newCell]=-1;
                        child[newCell-1]=-1;
                        child[newCell-2]=-1;
                        child[newCell-3]=-1;

                        //inserisco vecchia particella
                        childPath=0;
                        double down2=down,up2=up,left2=left,right2=right;

                        if(x[cell]<=0.5*(left2+right2)){
                            //+2
                            childPath +=2;
                            right2 = 0.5*(left2+right2);
                        }else{
                            left2 = 0.5*(left2+right2);
                        }

                        if(y[cell]>0.5*(up2+down2)){
                            //+1
                            childPath +=1;
                            down2 = 0.5*(up2+down2);
                        }else{
                            up2 = 0.5*(up2+down2);
                        }

                        //mass
                                                                                        printf("move lock:%d id:%d d %f, u %f, r %f, l %f\n",newCell-childPath,cell,down2,up2,right2,left2);
                        child[newCell-childPath]=cell;

                        //nuova particella
                        childPath=0;
                        father = newCell;
                        if(x[body]<=0.5*(left+right)){
                            //+2
                            childPath +=2;
                            right = 0.5*(left+right);
                        } else {
                            //+0
                            left = 0.5*(left+right);
                        }
                        if(y[body]>0.5*(up+down)){
                            //+1
                            childPath +=1;
                            down = 0.5*(up+down);
                        }else{
                            //+0
                            up = 0.5*(up+down);
                        }

                        cell=child[newCell-childPath];
                        //gestione doppio salto
                        child[newCell-childPath]=-2;

                        __threadfence();
                        child[lock]=newCell;

                        lock= newCell-childPath;

                    }
                                                                                        printf("lock:%d id:%d d %f, u %f, r %f, l %f\n",lock,body,down,up,right,left);
                    child[lock]=body;
                    finish=true;
                }
            }
            //__syncthreads();
        }
        cell = child[father - childPath];
    }
                                                                                //printf("%d",cell);
}

// funzione kernel per inizializzare la variabile globale puntatore
__global__ void setPointer(int num)
{
    pPointer = num-5;
}

// funzioni host

void getInput(FILE *file)
{
    x = (double *)malloc(sizeof(double) * numberBody);
    y = (double *)malloc(sizeof(double) * numberBody);
    m = (double *)malloc(sizeof(double) * numberBody);

    velX = (double *)malloc(sizeof(double) * numberBody);
    velY = (double *)malloc(sizeof(double) * numberBody);
    forceX = (double *)malloc(sizeof(double) * numberBody);
    forceY = (double *)malloc(sizeof(double) * numberBody);
    // prendo i dati per tutti i corpi
    printf("\n");
    for (int i = 0; i < numberBody; i++)
    {
        // prendo i dati dal file

        fscanf(file, "%lf%lf%lf%lf%lf", &x[i], &y[i], &m[i], &velX[i], &velY[i]);

        // imposto le forze iniziali a zero
        forceX[i] = 0;
        forceY[i] = 0;
        printf("particle %d xPos= %e, yPos= %e, mass= %e, forceX= %e, forceY= %e, velX= %e, velY= %e\n", i, x[i], y[i], m[i], forceX[i], forceY[i], velX[i], velY[i]);
    }
    printf("\n");
    // chiudo il file
    fclose(file);
}

FILE *initial()
{

    // mi apro il file in lettura
    FILE *file = fopen(fileInput, "r");
    // prendo il seed
    fscanf(file, "%d", &seed);
    printf("\n");
    printf("seed: %d\n", seed);
    // prendo il numero di corpi
    fscanf(file, "%d", &numberBody);
    printf("numero particelle: %d\n", numberBody);
    // calcolo max cell offset
    //maxCells = ((numberBody * 2 + 50) * 4);
    maxCells = ((numberBody * 2 + 12000) * 4);
    return file;
}

void printerTree(int* array, int state, int max,int point){
    if(state==0){
        int counter=0;
        for(int i=point;i>=0;i--){
            printf("%d , ",array[i]);
            counter++;
            if(counter%4==0){
                printf("(%d)\n",i);
            }
        }
        printf("  %d count %d",point,counter);
        printf("\n\n");
        printf("0\n");
        printerTree(array,state+1,max,point);
        printf("1\n");
        printerTree(array,state+1,max,point-1);
        printf("2\n");
        printerTree(array,state+1,max,point-2);
        printf("3\n");
        printerTree(array,state+1,max,point-3);
        return;
    }

    for(int i=0;i<state;i++){
        printf("\t");
    }
    if(array[point]<-1){
        printf("error");
        return;        
    }
    //printf("%d numero",array[point]);
    if(array[point]<max){
        if(array[point]==-1){
            printf("void\n");
        }else{
            printf("%d ",point);
            printf("point: %d\n",array[point]);
        }
        return;
    }
    printf("0\n");
    printerTree(array,state+1,max,array[point]);
    for(int i=0;i<state;i++){
        printf("\t");
    }
    printf("1\n");
    printerTree(array,state+1,max,array[point]-1);
    for(int i=0;i<state;i++){
        printf("\t");
    }
    printf("2\n");
    printerTree(array,state+1,max,array[point]-2);
    for(int i=0;i<state;i++){
        printf("\t");
    }
    printf("3\n");
    printerTree(array,state+1,max,array[point]-3);

}

void compute(int time)
{
    //printf("entro in compute\n");
    double *xP, *yP, up=maxSize, down=-maxSize, left=-maxSize, right=maxSize;
    int *child;

    //printf("inizio l'allocazione su device: \n");
    //printf("\n");
    hipMalloc((void **)&xP, sizeof(double) * numberBody);
    //printf("malloc 0 funzionante\n");
    gpuErrchk(hipMalloc((void **)&yP, sizeof(double) * numberBody));
    //printf("malloc 1 funzionante\n");
    gpuErrchk(hipMalloc((void **)&child, sizeof(int) * maxCells * 4));
    //printf("malloc 2 funzionante\n");
    //printf("\n");
    // copio array delle posizioni x e y delle particelle
    hipMemcpy(xP, x, sizeof(double) * numberBody, hipMemcpyHostToDevice);
    //printf("array x particelle copiato \n");
    hipMemcpy(yP, y, sizeof(double) * numberBody, hipMemcpyHostToDevice);
    //printf("array y particelle copiato \n");
    //printf("\n");
    // setto array dei figli a -1 (null)
    // cudaMemset(&child, -1, sizeof(int)*maxCells);
    hipMemset(&child[ maxCells - 1], -1, sizeof(int));
    hipMemset(&child[ maxCells - 2], -1, sizeof(int));
    hipMemset(&child[ maxCells - 3], -1, sizeof(int));
    hipMemset(&child[ maxCells - 4], -1, sizeof(int));
    //printf("array childs inizializzato \n");
    //printf("\n");
    // invoco la funzione per settarre la variabile puntatore globale nel device
    setPointer<<<1, 1>>>(maxCells);
    //printf("puntatore settato \n");
    //printf("\n");
    gpuErrchk(hipDeviceSynchronize());
    //printf("sincronizzo kernel");
    //printf("\n");

    // eseguo funzioni cuda
    for (int i = 0; i < time; i++)
    {

        // funzione che genera l'albero
        createTree<<<4, 1>>>(xP, yP, up, down, left, right, child, maxCells-1, numberBody);
        hipDeviceSynchronize();
        printf("albero generato, e kernel sincronizzati \n");
        int* childH=(int*) malloc( sizeof(int) * maxCells * 4);

        hipMemcpy(childH,child,sizeof(int) * maxCells * 4,hipMemcpyDeviceToHost);
        printerTree(childH,0,numberBody,maxCells-1);

        // calculateCenterMass<<<?>>>(?);
        // cudaDeviceSynchronize();
        // calculateMove<<<?>>>(?);
        // cudaDeviceSynchronize();
    }

    // libero memoria
    hipFree(child);
    hipFree(xP);
    hipFree(yP);
    //printf("memoria liberata sul device \n");
}

// stampa le particelle
void printer()
{
    printf("\n");
    for (int i = 0; i < numberBody; i++)
    {
        printf("particle %d xPos= %e, yPos= %e, mass= %e, forceX= %e, forceY= %e, velX= %e, velY= %e\n", i, x[i], y[i], m[i], forceX[i], forceY[i], velX[i], velY[i]);
    }

    printf("\n");
}
// stampa i risultati su solution.txt
void printerFile(){
    FILE* solution=fopen("solution.txt","w");
    for (int i = 0; i < numberBody; i++)
    {
        fprintf(solution,"%e,%e,%e,%e,%e,%e,%e\n", x[i], y[i], m[i], forceX[i], forceY[i], velX[i], velY[i]);
    }
    fclose(solution);
}

int main()
{
    // avvio getInput
    getInput(initial());
    // avvio compute
    //printf("avvio compute\n");
    compute(maxTime);
    // stampo i risultati del calcolo
    printer();
}