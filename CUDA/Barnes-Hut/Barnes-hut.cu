
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int numberBody, seed, maxTime = 3;
char fileInput[] = "../../Generate/particle.txt";

__constant__ double G = 6.67384E-11; // costante gravitazione universale
__constant__ double THETA = 0.5; // thetha per il calcolo delle forze su particell
__device__ int ppointer;


double maxSize = 6.162025e+070;
// double maxSize = 100;
// int count = 0;
double *x,*y,*m;
 //&p1[i].x, &p1[i].y, &p1[i].mass, &p1[i].velX, &p1[i].velY

//struct particella
typedef struct particle
{
    double x;      // posizione x
    double y;      // posizione y
    double mass;   // massa
    double forceX; // forza applicata alla particella sull' asse x
    double forceY; // forza applicata alla particella sull' asse y
    double velX;   // velocità sull' asse x
    double velY;   // velocità sull' asse y
    
} particle;

__device__ int findCell(int x,int y){
    printf("ppointer:%d\n",ppointer);
}

__global__ void createTree(double* xP,double* yP,double* up,double* down,double* left,double* right,int* child){
    
    int id=threadIdx.x+blockDim.x*blockIdx.x;
    int cell=findCell(xP[id],yP[id]);
}

__global__ void setppointer(int num){
    ppointer=num;
}

int statGPU() {
    int numberGPU;
    hipGetDeviceCount(&numberGPU);
    if(numberGPU<1){
        printf("non sono state rilevate GPU adeguate per esegiure il programma");
        exit(1);
    }

    hipDeviceProp_t pr;
    hipGetDeviceProperties(&pr,0);//thread per blocco 877
    int f = pr.sharedMemPerBlock/sizeof(particle); //massima dim memoria per blocco/grandezza struct particella 
    //printf("\n%d\n",f);

    if(pr.maxThreadsPerMultiProcessor%f){

        int h=pr.maxThreadsPerMultiProcessor;

        while (h>f)
        {
            h=h/2;
        }
        
        f=h;
    }
    printf("\n%d\n",f);
    return f;
}

void printerFile(particle *p1)
{
    FILE* solution=fopen("solution.txt","w");
    for (int i = 0; i < numberBody; i++)
    {
        fprintf(solution,"%e,%e,%e,%e,%e,%e,%e\n", p1[i].x, p1[i].y, p1[i].mass, p1[i].forceX, p1[i].forceY, p1[i].velX, p1[i].velY);
    }
    fclose(solution);
}

void printer(particle *p1)
{
    for (int i = 0; i < numberBody; i++)
    {
        printf("particle xPos= %e, yPos= %e, mass= %e, forceX= %e, forceY= %e, velX= %e, velY= %e\n", p1[i].x, p1[i].y, p1[i].mass, p1[i].forceX, p1[i].forceY, p1[i].velX, p1[i].velY);
    }
}

// calcolo il movimento delle particelle nel tempo richiesto
void compute(int time, particle *p1)
{
    /*
    int thread=statGPU();
    int block=(numberBody/thread)+1;
    */
    //int sizeTree=numberBody*2+12000;

    double *xP,*yP,*up,*down,*left,*right;
    int *child;
    
    // allocazione della memoria a device
    // gpuErrchk(); da aggiungere
    //cudaGetLastError
    printf("ciao\n");
    hipMalloc((void**)&xP,sizeof(double)*numberBody);
    printf("ciao\n");
    hipMalloc((void**)&yP,sizeof(double) * numberBody);
    hipMalloc((void**)&up,sizeof(double));
    hipMalloc((void**)&down,sizeof(double));
    hipMalloc((void**)&left,sizeof(double));
    hipMalloc((void**)&right,sizeof(double));
    hipMalloc((void**)&child,sizeof(int)*(numberBody*2+12000)*4);    
    
    hipMemcpy(xP,p1,sizeof(double) * numberBody,hipMemcpyHostToDevice);
    hipMemcpy(yP,p1,sizeof(double) * numberBody,hipMemcpyHostToDevice);
    hipMemset(up,maxSize,sizeof(double));
    hipMemset(down,-maxSize,sizeof(double));
    hipMemset(left,-maxSize,sizeof(double));
    hipMemset(right,maxSize,sizeof(double));
    hipMemset(&child[((numberBody*2+12000)*4)-1],-1,sizeof(int));

    setppointer<<<1,1>>>(((numberBody*2+12000)*4)-1);
    hipDeviceSynchronize();
    
    for(int i=0;i<time;i++){
        
        createTree<<<4,1>>>(xP,yP,up,down,left,right,child);
        hipDeviceSynchronize();
        //calculateCenterMass<<<?>>>(?);
        //cudaDeviceSynchronize();
        //calculateMove<<<?>>>(?);
        //cudaDeviceSynchronize();
                                                                                            //printf("\ncambio\n");
    }
    hipFree(xP);
    hipFree(yP);
    hipFree(up);
    hipFree(down);
    hipFree(left);
    hipFree(right);
    hipFree(child);

}

// popolo l'array con le particelle nel file
void getInput(FILE *file, particle *p1)
{
    // prendo i dati per tutti i corpi
    for (int i = 0; i < numberBody; i++)
    {
        // prendo i dati dal file
        fscanf(file, "%lf%lf%lf%lf%lf", &p1[i].x, &p1[i].y, &p1[i].mass, &p1[i].velX, &p1[i].velY);
        // imposto le forze iniziali a zero
        x[i]= p1[i].x;
        y[i]= p1[i].y;
        m[i]= p1[i].mass;
        p1[i].forceX = 0;
        p1[i].forceY = 0;
        //printf("particle xPos= %e, yPos= %e, mass= %e, forceX= %e, forceY= %e, velX= %e, velY= %e\n", p1[i].x, p1[i].y, p1[i].mass, p1[i].forceX, p1[i].forceY, p1[i].velX, p1[i].velY);
    }
    // chiudo il file
    fclose(file);
}

// aprire il file e prendere i primi valori (seed e numero di corpi)
FILE *initial()
{
    // mi apro il file in lettura
    FILE *file = fopen(fileInput, "r");
    // prendo il seed
    fscanf(file, "%d", &seed);
    printf("%d\n", seed);
    // prendo il numero di corpi
    fscanf(file, "%d", &numberBody);
    printf("%d\n", numberBody);
    return file;
}

int main()
{
    // apro il file dove si trovano tutte le particelle
    FILE *file = initial();
    // alloco memoria per variabili host
    particle *p1 = (particle*) malloc(sizeof(particle) * numberBody);
    x= (double*) malloc(sizeof(int)*numberBody);
    y= (double*) malloc(sizeof(int)*numberBody);
    m= (double*) malloc(sizeof(int)*numberBody);
    //inizializzo array di indirizzi child
    // popolo gli array
    getInput(file, p1);

    // calcolo il movimento delle particelle nel tempo richiesto
    compute(maxTime, p1);
    printf("\n");
    //printer(p1);

    //printerFile(p1);
    fclose(file);
    free(p1);
    exit(1);
}