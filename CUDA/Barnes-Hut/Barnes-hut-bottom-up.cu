
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>

// costanti e variabili host
int maxCells, numberBody, seed, maxTime = 10;
char fileInput[] = "../../Generate/particle.txt";
double *x, *y, *m, *velX, *velY, *forceX, *forceY;
int error_h=0;

hipDeviceProp_t pr;

// costanti e variabili gpu
__device__ const double G = 6.67384E-11; // costante gravitazione universale
__device__ const double THETA = 0;    // theta per il calcolo delle forze su particell
__device__ const int stackSize = 24;     // size dello stack per la gestione della ricorsione e per la pila delle profondità
__device__ const int blockSize = 256;    // dimensione dei bocchi, usata per gestire le memorie shared
__device__ int pPointer;                 // puntatore alla prima cella libera dell' array delle celle
__device__ const int deltaTime = 1;      // delta time
__device__ int error=0;


///////////////////////////////////////////GPU ERRORCHECK///////////////////////////////////////////////////////////////
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}
__device__ int h = 0;
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// funzioni gpu

__global__ void calculateMovement(double *xP, double *yP, double *mP,double *forceX, double *forceY, double *velX, double *velY,int numBody){

    int body = threadIdx.x + blockDim.x * blockIdx.x;

    if(body>=numBody){
        return;
    };

    xP[body] += deltaTime * velX[body];
    yP[body] += deltaTime * velY[body];
    velX[body] += deltaTime / mP[body] * forceX[body];
    velY[body] += deltaTime / mP[body] * forceY[body];
}

__global__ void calculateForce(int *child, double *xP, double *yP, double *mP, int point, int numBody, double *forceX, double *forceY, double *left, double *right)
{
    int body = threadIdx.x + blockDim.x * blockIdx.x;

    if(body>=numBody){
        return;
    };
    
    double size = (*right - *left);
    double forceXb = forceX[body];
    double forceYb = forceY[body];
    double xPb = xP[body];
    double yPb = yP[body];
    double mPb = mP[body];

    
    int cell=point;
    int pre=cell;
    for (int i = 1; i < 4; i++)
    {
        if(cell==-1){
            cell=child[pre-i];
        }else{
            break;
        }
    }

    pre=cell;
    
       

    // ciclo finchè ho filgi da analizzare, carico il figlio e aggiorno lo stack pointer
    while (true)
    {        
                                                                                                                                        //printf("cell: %d pre:%d\n",cell,pre);
        if (cell==-1)
        {
            break;
        }
        if (pre==cell)
        {
                                                                                                                                        //printf("analise: %d\n",cell);
            cell=child[pre];
            
            for (int i = 1; i < 4; i++)
            {
                                                                                                                                        //printf("add: %d\n",cell);
                if(cell==-1){
                    cell=child[pre-i];
                }else{
                    break;
                }
            }

                                                                                                                                        //printf("add2: %d\n",cell);

            pre=cell;

            if(cell==-1){
                break;
            }

        }else{

            //printf("%d , %d\n",cell, pre);
            int i;
            for (i = 0; i < 4; i++)
            {
                //printf("cella: %d\n",child[cell-i]);
                if (child[cell-i]==pre)
                {
                    //printf("cella: %d",cell);
                    pre=cell;
                    break;
                }
            }

            if(i==3){
                cell=child[pre-4];
                continue;
            }
            
            i++;
            cell=child[pre-i];
            i++;
            
            for(;i<4;i++){
                if(cell==-1){
                    cell=child[pre-i];
                }else{
                    break;
                }
            }

                                                                                                                                    //printf("cell: %d pre:%d\n",cell,pre);

            if(cell==-1){
                cell=child[pre-4];
                continue;
            }

            pre=cell;
        }
        
        double dist = sqrtf(pow(xPb - xP[cell], 2) + pow(yPb - yP[cell], 2));

        // controllo di non star confrontando la particella con se stessa
        if (dist == 0)
        {
            //printf("caza %d\n",child[pre]);
            cell=child[pre];
            continue;
        }

        // se sto guardando una particella calcolo le forze
        if (cell < numBody)
        {
                                                                                    // printf("size: %e\n",(size/pow(2,depth)));
            double xDiff = xPb - xP[cell];                             // calcolo la distanza tra la particella 1 e la 2
            double yDiff = yPb - yP[cell];                             // (il centro di massa del nodo = particella)
            double cubeDist = dist * dist * dist;                           // elevo al cubo la distanza e applico la formula di newton
            forceXb -= ((G * mPb * mP[cell]) / cubeDist) * xDiff; // per il calcolo della forza sui 2 assi
            forceYb -= ((G * mPb * mP[cell]) / cubeDist) * yDiff;

            cell=child[pre];

            continue;
                                                                                    //printf("dist:%e mass:%e xPcell: %e cell: %d\n",mPb,mP[cell],yP[cell],cell);
                                                                                    // printf("body %d, cell %d\n",body,cell);
        }
            // se va oltre il THETA calcolo approssimo, usiamo solo la x per il calcolo del tetha,  \
                (si potrebbe usare un abs max, per vedere chi è il massimo tra x e y) 
        if (((size / pow(2, child[cell-5])) / dist < THETA))
        {   
            double xDiff = xPb - xP[cell];                                 // calcolo la distanza tra la particella 1 e la 2
            double yDiff = yPb - yP[cell];                                 // (il centro di massa del nodo = particella)
            double cubeDist = dist * dist * dist;                          // elevo al cubo la distanza e applico la formula di newton
            forceXb -= ((G * mPb * mP[cell]) / cubeDist) * xDiff;          // per il calcolo della forza sui 2 assi
            forceYb -= ((G * mPb * mP[cell]) / cubeDist) * yDiff;

            cell=child[pre-4];
                                                                                //printf("teta %d, %d\n",cell,pre);
                                                                                // printf("body %d, size %d",body,((G * mP[body] * mP[cell]) / cubeDist) * xDiff);
        }
    }

                                                                                //printf("body %d, X %e, Y %e\n",body,forceX[body],forceY[body]);
    // aggiorno i valori delle particelle relative al delta-time
    forceX[body] = forceXb;
    forceY[body] = forceYb;
}
// setto la x e y iniziali
__global__ void initialPosition(double *up, double *down, double *left, double *right, double x,double y){
    
    *up=y;
    *down=y;
    *left=x;
    *right=x;
}
// calcolo la bounding box delle particelle, applicando tecniche di riduzione gpu
__global__ void boundingBox(double *xP, double *yP, int numBody, double *up, double *down, double *left, double *right, int *lock)
{

    // id del body gestito da i vari blocchi
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    // controllo se ho una particella da controllare
    if (id >= numBody)
    {
        return;
    }

    // creiamo due tipi variabili, una locale e un array (shared) per ogniuno dei 4 valori
    float xMin = xP[id];
    float xMax = xP[id];
    float yMin = yP[id];
    float yMax = yP[id];

    __shared__ float leftCache[blockSize];
    __shared__ float rightCache[blockSize];
    __shared__ float upCache[blockSize];
    __shared__ float downCache[blockSize];

    int offset = stride;

    // finche mi trovo tra le particelle, cerco i valori minimi e massimi
    while (id + offset < numBody)
    {
        xMin = fminf(xMin, xP[id + offset]);
        xMax = fmaxf(xMax, xP[id + offset]);
        yMin = fminf(yMin, yP[id + offset]);
        yMax = fmaxf(yMax, yP[id + offset]);
        offset += stride;
    }

    // salvo i valori nella memoria shared relativa al thread
    leftCache[threadIdx.x] = xMin;
    rightCache[threadIdx.x] = xMax;
    upCache[threadIdx.x] = yMax;
    downCache[threadIdx.x] = yMin;

    __syncthreads();

    // applico la riduzione dimezzando ogni volta i thread, ottimizzando l'utilizzo dei warp
    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (threadIdx.x < i)
        {   
            if(id + i<numBody){
                                                                                                                    //printf("indicatore di stocazzo %i\n",threadIdx.x + i);
                // confronto i valori vari con quelli in shared e li sovrascrivo con i rispettivi minimi e massimi
                leftCache[threadIdx.x] = fminf(leftCache[threadIdx.x], leftCache[threadIdx.x + i]);
                rightCache[threadIdx.x] = fmaxf(rightCache[threadIdx.x], rightCache[threadIdx.x + i]);
                upCache[threadIdx.x] = fmaxf(upCache[threadIdx.x], upCache[threadIdx.x + i]);
                downCache[threadIdx.x] = fminf(downCache[threadIdx.x], downCache[threadIdx.x + i]);
            }
        }
        __syncthreads();
        i /= 2;
    }
    
    // il thread 0, esegue la funzione di master e confronta il risultato con gli altri bloccchi
    if (threadIdx.x == 0)
    {
        // utiliziamo un mutex per accedere alla memoria globale evitando concorrenza
        while (atomicCAS(lock, 0, 1) != 0);

        *left = fminf(*left, leftCache[0]);
        *right = fmaxf(*right, rightCache[0]);
        *up = fmaxf(*up, upCache[0]);
        *down = fminf(*down, downCache[0]);

        atomicExch(lock, 0);
    }
}

// espandiamo la bounding box, per allinearci con i calcoli svolti nell'implementazione single thread
__global__ void boundingBoxExpander(double *up, double *down, double *left, double *right)
{
    *right = *right + 1;
    *left = *left - 1;
    *up = *up + 1;
    *down = *down - 1;

   /**right=20;
   *left=-20;
   *up =20;
   *down =-20;*/
                                                                        printf("\n\nBounding box: up: %e,down: %e,left: %e,right: %e\n\n",*up,*down,*left,*right);
}

//funzione di calcolo dei centri di massa
__global__ void calculateCenterMass(int* child,double* xP,double* yP,double* mP,double numBody){
    
    int body=threadIdx.x + blockDim.x * blockIdx.x;
    if(body>=numBody){
        return;
    }
    int cell=child[body];
    while (true){
        double mass=0;
        double mcX=0;
        double mcY=0;

        for(int i=0;i<4;i++){
            int childCell=child[cell-i];
            if(childCell==-1){
                continue;
            }
            double massCell=mP[childCell];
            if(massCell==0){
                return;
            }
            mass+=massCell;
            mcX+=massCell*xP[childCell];
            mcY+=massCell*yP[childCell];
        }
        xP[cell]=mcX/mass;
        yP[cell]=mcY/mass;
        mP[cell]=mass;
                                                                                //printf("cell %d, x %e, y %e, mass %e\n",cell,xP[cell],yP[cell],mass);
        cell=child[cell-4];
        if(cell==-1){
            return;
        }
    }
}

// funzione per la creazione dell'albero
__global__ void createTree(double* x, double* y,double* mass, double* upA, double* downA, double* leftA, double* rightA, int *child,int cell,int numBody)
{  
    // DA SISTEMARE DIOCAN
    double up=*upA, down = *downA, left=*leftA, right =*rightA;
    int body = threadIdx.x + blockDim.x * blockIdx.x;
    // uccido il thread che non deve inserire particelle
    if(body>numBody){
        return;
    }
    int father=cell;
    bool newBody=true;
    bool finish=false;
    int childPath;
    while(!finish){

        //se inserisco una nuova particella
        if(newBody){
            newBody = false;
            childPath = 0;

            //assegno i path ai figli
            if(x[body]<=0.5*(left+right)){
                //+2                            //    _________________________
                childPath +=2;                  //   |          3 |          1 |
                right = 0.5*(left+right);       //   |    (NW)    |    (NE)    |
            } else {                            //   |            |            |
                //+0                            //   |     -+     |     ++     |
                left = 0.5*(left+right);        //   |____________|____________|
            }                                   //   |          2 |          0 |
            if(y[body]>0.5*(up+down)){          //   |     --     |     +-     |
                //+1                            //   |            |            |
                childPath +=1;                  //   |    (SW)    |    (SE)    |
                down = 0.5*(up+down);           //   |____________|____________|
            }else{
                //+0
                up = 0.5*(up+down);
            }  
        }
        cell=child[father-childPath];
        // ciclo fino a che non trovo una foglia e assegno i path
        while(cell >= numBody){
            
            father = cell;
            childPath=0;
            if(x[body]<=0.5*(left+right)){
                //+2
                childPath +=2;
                right = 0.5*(left+right);
            } else {
                //+0
                left = 0.5*(left+right);
            }
            if(y[body]>0.5*(up+down)){
                //+1
                childPath +=1;
                down = 0.5*(up+down);
            }else{
                //+0
                up = 0.5*(up+down);
            }

            cell = child[father - childPath];
        }
                                                                                //printf("cell: %d\n",cell);
        //controllo se la cella è libera
        if (cell != -2){
            int lock=father-childPath;
                                                                                //printf("cell2: %d\n",cell);
            //blocco la cella per lavoraci, utilizzando una funzione atomica
            if(atomicCAS(&child[lock],cell,-2)==cell){
                if(cell == -1){
                                                                                //printf("lock:%d id:%d d %f, u %f, r %f, l %f\n",lock,body,down,up,right,left);
                    //child[body]=lock;
                    child[body]=father;
                    child[lock] = body;
                    finish=true;     
                }else{
                    while(cell>=0 && cell<numBody){

                        //scalo al prossimo indice con cella libera
                        int newCell = atomicAdd(&pPointer,-6);
                        if(newCell-6<numBody){
                            printf("\nNon ho spazio disponibile\n");
                            error=1;
                            return;
                        }
                        //assegno ai figli il valore -1, ovvero puntatore a null
                        child[newCell]=-1;
                        child[newCell-1]=-1;
                        child[newCell-2]=-1;
                        child[newCell-3]=-1;
                        child[newCell-4]=father;
                        child[newCell-5]=child[father-5]+1;
                        
                        //inserisco la vecchia particella
                        childPath=0;
                                                                                        //double down2=down,up2=up,left2=left,right2=right;

                        if(x[cell]<=0.5*(left+right)){
                            //+2
                            childPath +=2;
                            //right = 0.5*(left+right);
                        }else{
                            //left = 0.5*(left+right);
                        }
                        if(y[cell]>0.5*(up+down)){
                            //+1
                            childPath +=1;
                            //down = 0.5*(up+down);
                        }else{
                            //up = 0.5*(up+down);
                        }
                                                                                        //printf("move lock:%d id:%d d %f, u %f, r %f, l %f\n",newCell-childPath,cell,down2,up2,right2,left2);
                        //child[cell]=newCell-childPath;

                        child[cell]=newCell;
                        child[newCell-childPath]=cell;

                        //vedo dove inserire una nuova particella
                        childPath=0;
                        father = newCell;
                        if(x[body]<=0.5*(left+right)){
                            //+2
                            childPath +=2;
                            right = 0.5*(left+right);
                        } else {
                            //+0
                            left = 0.5*(left+right);
                        }
                        if(y[body]>0.5*(up+down)){
                            //+1
                            childPath +=1;
                            down = 0.5*(up+down);
                        }else{
                            //+0
                            up = 0.5*(up+down);
                        }
                        
                        cell=child[newCell-childPath];
                        
                        child[newCell-childPath]=-2;
                        

                        __threadfence();
                        child[lock]=newCell;

                        lock= newCell-childPath;

                    }
                                                                                        //printf("lock:%d id:%d d %f, u %f, r %f, l %f\n",lock,body,down,up,right,left);
                    //child[body]=lock;

                    child[body]=father;
                    child[lock]=body;
                    finish=true;
                }
            }
            //__syncthreads();
        }
        cell = child[father - childPath];

    }
                                                                                //printf("%d",cell);
}

// funzione kernel per inizializzare la variabile globale puntatore
__global__ void setPointer(int num)
{
    pPointer = num-7;
}

// funzioni host

void getInput(FILE *file)
{
    x = (double *)malloc(sizeof(double) * numberBody);
    y = (double *)malloc(sizeof(double) * numberBody);
    m = (double *)malloc(sizeof(double) * numberBody);
    velX = (double *)malloc(sizeof(double) * numberBody);
    velY = (double *)malloc(sizeof(double) * numberBody);
    forceX = (double *)malloc(sizeof(double) * numberBody);
    forceY = (double *)malloc(sizeof(double) * numberBody);

    // prendo i dati per tutti i corpi
    printf("\n");
    for (int i = 0; i < numberBody; i++)
    {
        // prendo i dati dal file
        fscanf(file, "%lf%lf%lf%lf%lf", &x[i], &y[i], &m[i], &velX[i], &velY[i]);

        // imposto le forze iniziali a zero
        forceX[i] = 0;
        forceY[i] = 0;
        printf("particle %d xPos= %e, yPos= %e, mass= %e, forceX= %e, forceY= %e, velX= %e, velY= %e\n",\
        i, x[i], y[i], m[i], forceX[i], forceY[i], velX[i], velY[i]);
    }
    printf("\n");
    // chiudo il file
    fclose(file);
}

FILE *initial()
{

    // mi apro il file in lettura
    FILE *file = fopen(fileInput, "r");
    // prendo il seed
    fscanf(file, "%d", &seed);
    printf("\n");
    printf("seed: %d\n", seed);
    // prendo il numero di corpi
    fscanf(file, "%d", &numberBody);
    printf("numero particelle: %d\n", numberBody);
    // calcolo max cell offset
    maxCells = ((numberBody * 2 + 50) * 4);
    //maxCells = ((numberBody * 2 + 12000) * 4);
    return file;
}

                                                                                                    __global__ void set0(int* child){
                                                                                                        child[pPointer-4]=0;
                                                                                                    }

__global__ void checkError(int* er){
    *er=error;
}

__global__ void resetArray(double *xP, double *yP, double *massP, int point)
{

    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    point -= 4;

    for (int i = point - (4 * id); i > pPointer; i -= (4 * stride))
    {

                                                            // printf("\n(%d) ",i);
        xP[i] = 0;
        yP[i] = 0;
        massP[i] = 0;
                                                            // printf("(%d)mass: %e x:%e y:%e\n",i,mP[i],xP[i],yP[i]);
    }
}

//funzione grafica per stampare l'albero creato da crateTree()
void printerTree(int* array, int state, int max,int point){
    if(state==0){
        int counter=0;
        printf("(%d) ",point);
        for(int i=point;i>=0;i--){
            printf("%d , ",array[i]);
            counter++;
            if(counter%6==0){
                if(array[i-5]==0){
                    break;
                }
                printf("\n(%d) ",i-1);
            }
        }
        printf("\n\nPosizione dei body: ");
        int counter2=max;
        for(int i=max-1;i>=0;i--){
            counter2--;
            printf("(%d) %d , ",counter2,array[i]);
        }
        printf("\n");
        return;
        printf("\n%d count %d",point,counter);
        printf("\n\n");
        printf("0\n");
        printerTree(array,state+1,max,point);
        printf("1\n");
        printerTree(array,state+1,max,point-1);
        printf("2\n");
        printerTree(array,state+1,max,point-2);
        printf("3\n");
        printerTree(array,state+1,max,point-3);
        return;
    }

    for(int i=0;i<state;i++){
        printf("\t");
    }
    if(array[point]<-1){
        printf("error");
        return;        
    }
    //printf("%d numero",array[point]);
    if(array[point]<max){
        if(array[point]==-1){
            printf("void\n");
        }else{
            printf("%d ",point);
            printf("point: %d\n",array[point]);
        }
        return;
    }
    printf("0\n");
    printerTree(array,state+1,max,array[point]);
    for(int i=0;i<state;i++){
        printf("\t");
    }
    printf("1\n");
    printerTree(array,state+1,max,array[point]-1);
    for(int i=0;i<state;i++){
        printf("\t");
    }
    printf("2\n");
    printerTree(array,state+1,max,array[point]-2);
    for(int i=0;i<state;i++){
        printf("\t");
    }
    printf("3\n");
    printerTree(array,state+1,max,array[point]-3);

}

// riporto i valori da kernel a host
void returnCuda(double *xP, double *yP, double *velXP, double *velYP, double *forceXP, double *forceYP)
{

    gpuErrchk(hipMemcpy(x, xP, sizeof(double) * numberBody, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(y, yP, sizeof(double) * numberBody, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(velX, velXP, sizeof(double) * numberBody, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(velY, velYP, sizeof(double) * numberBody, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(forceX, forceXP, sizeof(double) * numberBody, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(forceY, forceYP, sizeof(double) * numberBody, hipMemcpyDeviceToHost));
}

//funzione di esecuzione dei vari kernell
void compute(int time)
{
    int *er;
    hipMalloc((void**)&er,sizeof(int));

    double *xP, *yP, *massP;
    
    double *up, *down, *left, *right;
    int *child,*lock;

    double *forceXP, *forceYP, *velXP, *velYP;
    
        //variabili di ottimizzazione GPU
    int boundingNumBlocks=(numberBody/blockSize)+1;

    int preciseNumThread=pr.maxThreadsPerBlock;
    int preciseNumBlocks;
    int preciseNumBlockSize=(numberBody/blockSize)+1;

    if(numberBody<preciseNumThread){

        preciseNumBlocks = 1;
        preciseNumThread = numberBody;

    }else{

        if(pr.maxThreadsPerMultiProcessor%preciseNumThread!=0){

            preciseNumThread=pr.maxThreadsPerMultiProcessor/2;

        }

        preciseNumBlocks=(numberBody/preciseNumThread)+1;

    }

    //alloco la memoria dei vari parametrio sul device
    gpuErrchk(hipMalloc((void **)&xP, sizeof(double) * maxCells * 4));
    gpuErrchk(hipMalloc((void **)&yP, sizeof(double) * maxCells * 4));   
    gpuErrchk(hipMalloc((void **)&child, sizeof(int) * maxCells * 4));   
    gpuErrchk(hipMalloc((void **)&massP, sizeof(double) * maxCells * 4));
    gpuErrchk(hipMalloc((void **)&forceXP, sizeof(double) * numberBody));
    gpuErrchk(hipMalloc((void **)&forceYP, sizeof(double) * numberBody));
    gpuErrchk(hipMalloc((void **)&velXP, sizeof(double) * numberBody));
    gpuErrchk(hipMalloc((void **)&velYP, sizeof(double) * numberBody));
    
    gpuErrchk(hipMalloc((void **)&up, sizeof(double)));
    gpuErrchk(hipMalloc((void **)&down, sizeof(double)));
    gpuErrchk(hipMalloc((void **)&left, sizeof(double)));
    gpuErrchk(hipMalloc((void **)&right, sizeof(double)));
    gpuErrchk(hipMalloc((void **)&lock, sizeof(int)));
    // copio array delle posizioni x, y e masse delle particelle
    hipMemcpy(xP, x, sizeof(double) * numberBody, hipMemcpyHostToDevice);   
    hipMemcpy(yP, y, sizeof(double) * numberBody, hipMemcpyHostToDevice);  
    hipMemcpy(massP, m, sizeof(double) * numberBody, hipMemcpyHostToDevice);
    gpuErrchk(hipMemcpy(velXP, velX, sizeof(double) * numberBody, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(velYP, velY, sizeof(double) * numberBody, hipMemcpyHostToDevice));
                                                                                                int* childH=(int*) malloc( sizeof(int) * maxCells * 4);
    
    gpuErrchk(hipDeviceSynchronize());
    // eseguo funzioni cuda
    for (int i = 0; i < time; i++)
    {
        printf("%d\n",i);
        // invoco la funzione per settarre la variabile puntatore globale nel device
        setPointer<<<1,1>>>(maxCells);

        initialPosition<<<1,1>>>(up, down, left, right,x[0],y[0]);

        hipDeviceSynchronize();
        
        boundingBox<<<boundingNumBlocks, blockSize>>>(xP, yP, numberBody, up, down, left, right, lock);     //sizeBlock

        hipDeviceSynchronize();

        boundingBoxExpander<<<1, 1>>>(up, down, left, right);
        // setto array dei figli a -1 (null)
        hipMemset(&child[ maxCells - 1], -1, sizeof(int));
        hipMemset(&child[ maxCells - 2], -1, sizeof(int));
        hipMemset(&child[ maxCells - 3], -1, sizeof(int));
        hipMemset(&child[ maxCells - 4], -1, sizeof(int));
        hipMemset(&child[ maxCells - 5], -1, sizeof(int));
        hipMemset(&child[ maxCells - 6], 0, sizeof(int));

        // genero l'albero
        createTree<<<preciseNumBlocks, preciseNumThread>>>(xP, yP, massP, up, down, left, right, child, maxCells-1, numberBody);
        hipDeviceSynchronize();
        // sincronizzo i kernel a fine esecuzione

        checkError<<<1,1>>>(er);
        gpuErrchk(hipMemcpy(&error_h, er, sizeof(int), hipMemcpyDeviceToHost));
        if(error_h!=0){
            break;
        }
                                                                                                set0<<<1,1>>>(child);
                                                                                                hipMemcpy(childH,child,sizeof(int) * maxCells * 4,hipMemcpyDeviceToHost);
                                                                                                // ritorno l'albero a l'host per la stampa e lo stampo
                                                                                                printerTree(childH,0,numberBody,maxCells-1);
         
        
        // calcolo centri di massa
        calculateCenterMass<<<preciseNumBlocks, preciseNumThread>>>(child,xP,yP,massP,numberBody);
        hipDeviceSynchronize();

        calculateForce<<<preciseNumBlockSize, blockSize>>>(child, xP, yP, massP, maxCells - 1, numberBody, forceXP, forceYP, left, right);    //precisa sizeBlock
        hipDeviceSynchronize();

        checkError<<<1,1>>>(er);
        gpuErrchk(hipMemcpy(&error_h, er, sizeof(int), hipMemcpyDeviceToHost));
        if(error_h!=0){
            break;
        }
        
        // calcolo spostamento particelle
        calculateMovement<<<preciseNumBlocks,preciseNumThread>>>(xP, yP, massP,forceXP, forceYP, velXP, velYP, numberBody); //precisa
        hipDeviceSynchronize();

        resetArray<<<preciseNumBlocks, preciseNumThread>>>(xP, yP, massP, maxCells - 1);
        gpuErrchk(hipMemset(lock, 0, sizeof(int)));

        hipDeviceSynchronize();
        
    }
    returnCuda(xP, yP, velXP, velYP, forceXP, forceYP);
    // libero memoria
                                                                                                free(childH);
    hipFree(child);
    hipFree(xP);
    hipFree(yP);
    hipFree(massP);
    //printf("memoria liberata sul device \n");
}

// stampa le particelle
void printer()
{
    printf("\n");
    for (int i = 0; i < numberBody; i++)
    {
        printf("particle %d xPos= %e, yPos= %e, mass= %e, forceX= %e, forceY= %e, velX= %e, velY= %e\n", i, x[i], y[i], m[i], forceX[i], forceY[i], velX[i], velY[i]);
    }

    printf("\n");
}
// stampa i risultati su solution.txt
void printerFile(){
    FILE* solution=fopen("solution.txt","w");
    for (int i = 0; i < numberBody; i++)
    {
        fprintf(solution,"%e,%e,%e,%e,%e,%e,%e\n", x[i], y[i], m[i], forceX[i], forceY[i], velX[i], velY[i]);
    }
    fclose(solution);
}

int statGPU()
{
    int numberGPU;
    hipGetDeviceCount(&numberGPU);
    if (numberGPU < 1)
    {
        printf("Non sono state rilevate GPU Cuda per esegiure il programma");
        exit(1);
    } 

    hipGetDeviceProperties(&pr, 0);
}

int main()
{
    // verifico le stat della gpu e la sua presenza
    statGPU();
    // avvio getInput
    getInput(initial());
    // avvio compute
    compute(maxTime);
    // stampo i risultati del calcolo
    printer();
}